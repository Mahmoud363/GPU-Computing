#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<math.h>
#include<time.h>
#include<stdlib.h>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b, const int n)
{
    int i =blockIdx.x*blockDim.x+ threadIdx.x;
	if(i*2<n)
		c[i * 2] = a[i * 2] + b[i * 2];
	if((i*2+1)<n)
		c[i * 2 + 1] = a[i * 2 + 1] + b[i * 2 + 1];
}

int main()
{
	int arraySize;
	printf("Enter the size of your array: ");
	scanf("%d", &arraySize);
	int* a = (int*)malloc(arraySize * sizeof(int));

	 // if memory cannot be allocated
	 if (a == NULL)
	 {
		 printf("Error! memory not allocated.");
		 exit(0);
	 }
	 int* b = (int*)malloc(arraySize * sizeof(int));

	 // if memory cannot be allocated
	 if (b == NULL)
	 {
		 printf("Error! memory not allocated.");
		 exit(0);
	 }
	 int* c = (int*)malloc(arraySize * sizeof(int));

	 // if memory cannot be allocated
	 if (c == NULL)
	 {
		 printf("Error! memory not allocated.");
		 exit(0);
	 }
	srand(time(0));
	for (int i = 0; i < arraySize; i++)
	{
		a[i] = rand();
		b[i] = rand();
	}



    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	for (int i = 0; i < arraySize; i++)
	{
		printf("%d", c[i]);
		((i+1) % 20 == 0) ? printf("\n") : printf(" ");
	}
   

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    free(a);
    free(b);
    free(c);
    return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }
	dim3 gridSize(ceil(ceil(size/2.0) / 128.0), 1, 1);
	dim3 blockSize(128, 1, 1);
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<gridSize, blockSize>>>(dev_c, dev_a, dev_b, size);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
